#include "hip/hip_runtime.h"
/* Copyright 2025 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "bfloat16.h"
#include "hopper/linear_hopper.cuh"
#include "hopper/multitoken_paged_attention_hopper.cuh"
#include "hopper/norm_linear_hopper.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

using kernel::linear_kernel_hopper;
using kernel::multitoken_paged_attention_hopper_impl;
using kernel::norm_linear_kernel_hopper;
using bfloat16 = type::bfloat16_t;

template <typename T,
          int BATCH_SIZE,
          int OUTPUT_SIZE,
          int REDUCTION_SIZE,
          typename TMA_A,
          typename TMA_B,
          typename TMA_RESIDUAL,
          typename TMA_OUT,
          int Kstages = 2>
__global__ __launch_bounds__(256, 1) void linear_kernel_hopper_wrapper(
    const __grid_constant__ TMA_A tma_a,
    const __grid_constant__ TMA_B tma_b,
    const __grid_constant__ TMA_RESIDUAL tma_residual,
    const __grid_constant__ TMA_OUT tma_out) {
  linear_kernel_hopper<T,
                       BATCH_SIZE,
                       OUTPUT_SIZE,
                       REDUCTION_SIZE,
                       Kstages,
                       TMA_A,
                       TMA_B,
                       TMA_OUT>(tma_a, tma_b, tma_residual, tma_out);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_linear_hopper(void *input_ptr,
                          void *weight_ptr,
                          void *residual_ptr,
                          void *output_ptr) {

  constexpr int B = 3;
  constexpr int M = 3;
  constexpr int S = 3;

  constexpr int TMA_CP_ASYNC_SIZE =
      64; // note that if swizzle 128 is used, 64 is maximal cp size
  constexpr int TILE_SIZE =
      128; // we should modify this param if we want larger tile size
  constexpr int TMA_CP_ASYNC_REPEAT_COL =
      (TILE_SIZE + TMA_CP_ASYNC_SIZE - 1) / TMA_CP_ASYNC_SIZE;
  constexpr int OUTPUT_REPEAT_COL =
      (OUTPUT_SIZE + TMA_CP_ASYNC_SIZE - 1) / TMA_CP_ASYNC_SIZE;

  constexpr int OUTPUT_TMA_CP_SIZE = OUTPUT_SIZE < 64 ? OUTPUT_SIZE : 64;

  using TMA_A = kernel::tma::tma_2d<bfloat16,
                                    B,
                                    M,
                                    S,
                                    BATCH_SIZE,
                                    REDUCTION_SIZE,
                                    BATCH_SIZE,
                                    TMA_CP_ASYNC_SIZE,
                                    1,
                                    TMA_CP_ASYNC_REPEAT_COL,
                                    true>;
  using TMA_B = kernel::tma::tma_2d<bfloat16,
                                    B,
                                    M,
                                    S,
                                    OUTPUT_SIZE,
                                    REDUCTION_SIZE,
                                    OUTPUT_SIZE,
                                    TMA_CP_ASYNC_SIZE,
                                    1,
                                    TMA_CP_ASYNC_REPEAT_COL,
                                    true>;
  using TMA_RESIDUAL = kernel::tma::tma_2d<bfloat16,
                                           0,
                                           0,
                                           0,
                                           BATCH_SIZE,
                                           OUTPUT_SIZE,
                                           BATCH_SIZE,
                                           OUTPUT_TMA_CP_SIZE,
                                           1,
                                           OUTPUT_REPEAT_COL,
                                           true>;

  using TMA_OUT = kernel::tma::tma_2d<bfloat16,
                                      0,
                                      0,
                                      0,
                                      BATCH_SIZE,
                                      OUTPUT_SIZE,
                                      BATCH_SIZE,
                                      OUTPUT_TMA_CP_SIZE,
                                      1,
                                      OUTPUT_REPEAT_COL,
                                      true>;
  TMA_A tma_a(input_ptr);
  TMA_B tma_b(weight_ptr);
  TMA_RESIDUAL tma_residual(residual_ptr);
  TMA_OUT tma_out(output_ptr);

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(256, 1, 1);
  size_t smem_size = 88888;
  hipFuncSetAttribute(reinterpret_cast<const void*>(linear_kernel_hopper_wrapper<T),
                                                    BATCH_SIZE,
                                                    OUTPUT_SIZE,
                                                    REDUCTION_SIZE,
                                                    TMA_A,
                                                    TMA_B,
                                                    TMA_RESIDUAL,
                                                    TMA_OUT>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

#ifndef MIRAGE_PROFILE_HOPPER
  linear_kernel_hopper_wrapper<T,
                               BATCH_SIZE,
                               OUTPUT_SIZE,
                               REDUCTION_SIZE,
                               TMA_A,
                               TMA_B,
                               TMA_RESIDUAL,
                               TMA_OUT>
      <<<grid_dim, block_dim, smem_size>>>(tma_a, tma_b, tma_residual, tma_out);
#else

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  constexpr int WARMUP_RUNS = 16;
  constexpr int BENCHMARK_RUNS = 1000;

  printf("=== Kernel Performance Profiling ===\n");

  for (int i = 0; i < WARMUP_RUNS; i++) {
    linear_kernel_hopper_wrapper<T,
                                 BATCH_SIZE,
                                 OUTPUT_SIZE,
                                 REDUCTION_SIZE,
                                 TMA_A,
                                 TMA_B,
                                 TMA_RESIDUAL,
                                 TMA_OUT><<<grid_dim, block_dim, smem_size>>>(
        tma_a, tma_b, tma_residual, tma_out);
  }
  hipDeviceSynchronize(); // Wait for all warmup runs to complete

  printf("Running %d benchmark iterations...\n", BENCHMARK_RUNS);

  float *iteration_times = new float[BENCHMARK_RUNS];
  float total_time_ms = 0.0f;
  float min_time_ms = FLT_MAX;
  float max_time_ms = 0.0f;

  for (int i = 0; i < BENCHMARK_RUNS; i++) {
    hipEventRecord(start);
    linear_kernel_hopper_wrapper<T,
                                 BATCH_SIZE,
                                 OUTPUT_SIZE,
                                 REDUCTION_SIZE,
                                 TMA_A,
                                 TMA_B,
                                 TMA_RESIDUAL,
                                 TMA_OUT><<<grid_dim, block_dim, smem_size>>>(
        tma_a, tma_b, tma_residual, tma_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float iteration_time_ms;
    hipEventElapsedTime(&iteration_time_ms, start, stop);

    total_time_ms += iteration_time_ms;
  }

  float avg_time_ms = total_time_ms / BENCHMARK_RUNS;

  printf("\n=== Performance Results ===\n");
  printf("Configuration:\n");
  printf("  BATCH_SIZE=%d, OUTPUT_SIZE=%d, REDUCTION_SIZE=%d\n",
         BATCH_SIZE,
         OUTPUT_SIZE,
         REDUCTION_SIZE);
  printf(" TILE SIZE: %d\n", TILE_SIZE);
  printf("  Average: %.3f ms\n", avg_time_ms);

  printf("===============================\n");

  delete[] iteration_times;
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
#endif

#define DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(                            \
    BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE)                                   \
  case REDUCTION_SIZE:                                                         \
    launch_linear_hopper<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(   \
        input_ptr, weight_ptr, residual_ptr, output_ptr);                      \
    break;

#define DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE(BATCH_SIZE, OUTPUT_SIZE)         \
  switch (input.size(1)) {                                                     \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 64)    \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 128)   \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 256)   \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 512)   \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 3072)  \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, 4096)  \
    default:                                                                   \
      printf("Unsupported reduction size in test: %zu\n", input.size(1));      \
      break;                                                                   \
  }

#define DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE)       \
  case OUTPUT_SIZE:                                                            \
    DISPATCH_LINEAR_HOPPER_REDUCTION_SIZE(BATCH_SIZE, OUTPUT_SIZE)             \
    break;

#define DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE(BATCH_SIZE)                         \
  switch (output.size(1)) {                                                    \
    DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 16)                    \
    DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 32)                    \
    DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 64)                    \
    default:                                                                   \
      printf("Unsupported output size in test: %zu\n", output.size(1));        \
      break;                                                                   \
  }

#define DISPATCH_LINEAR_HOPPER_BATCH_SIZE_CASE(BATCH_SIZE)                     \
  case BATCH_SIZE:                                                             \
    DISPATCH_LINEAR_HOPPER_OUTPUT_SIZE(BATCH_SIZE)                             \
    break;

  void linear_kernel(torch::Tensor input,
                     torch::Tensor weight,
                     torch::Tensor residual,
                     torch::Tensor output) {

    void *input_ptr = input.data_ptr();
    void *weight_ptr = weight.data_ptr();
    void *residual_ptr = residual.data_ptr();
    void *output_ptr = output.data_ptr();

    switch (input.size(0)) {
      //  DISPATCH_LINEAR_HOPPER_BATCH_SIZE_CASE(16)
      DISPATCH_LINEAR_HOPPER_BATCH_SIZE_CASE(64)
      default:
        printf("Unsupported batch size in test: %zu\n", output.size(0));
        break;
    }

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
  }

  // norm linear
  template <typename T,
            int BATCH_SIZE,
            int OUTPUT_SIZE,
            int REDUCTION_SIZE,
            typename TMA_INPUT,
            typename TMA_NORM_WEIGHT,
            typename TMA_LINEAR_WEIGHT,
            typename TMA_OUT,
            int Kstages = 2>
  __global__ __launch_bounds__(256, 1) void norm_linear_kernel_hopper_wrapper(
      const __grid_constant__ TMA_INPUT tma_input,
      const __grid_constant__ TMA_NORM_WEIGHT tma_norm_weight,
      const __grid_constant__ TMA_LINEAR_WEIGHT tma_linear_weight,
      const __grid_constant__ TMA_OUT tma_out,
      float eps) {
    norm_linear_kernel_hopper<T,
                              BATCH_SIZE,
                              OUTPUT_SIZE,
                              REDUCTION_SIZE,
                              Kstages,
                              TMA_INPUT,
                              TMA_NORM_WEIGHT,
                              TMA_LINEAR_WEIGHT,
                              TMA_OUT>(
        tma_input, tma_norm_weight, tma_linear_weight, tma_out, eps);
  }

  template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
  void launch_norm_linear_hopper(void *input_ptr,
                                 void *norm_weight_ptr,
                                 void *weight_ptr,
                                 void *output_ptr,
                                 float eps) {

    constexpr int B = 3;
    constexpr int M = 3;
    constexpr int S = 3;

    constexpr int TILE_SIZE = 64;

    using TMA_INPUT = kernel::tma::tma_2d<bfloat16,
                                          B,
                                          M,
                                          S,
                                          BATCH_SIZE,
                                          REDUCTION_SIZE,
                                          BATCH_SIZE,
                                          TILE_SIZE>;
    using TMA_NORM_WEIGHT = kernel::tma::tma_2d<bfloat16,
                                                B,
                                                M,
                                                S,
                                                BATCH_SIZE,
                                                REDUCTION_SIZE,
                                                BATCH_SIZE,
                                                TILE_SIZE>;
    using TMA_LINEAR_WEIGHT = kernel::tma::tma_2d<bfloat16,
                                                  B,
                                                  M,
                                                  S,
                                                  OUTPUT_SIZE,
                                                  REDUCTION_SIZE,
                                                  OUTPUT_SIZE,
                                                  TILE_SIZE>;

    using TMA_OUT = kernel::tma::tma_2d<bfloat16,
                                        0,
                                        0,
                                        0,
                                        BATCH_SIZE,
                                        OUTPUT_SIZE,
                                        BATCH_SIZE,
                                        OUTPUT_SIZE>;

    TMA_INPUT tma_input(input_ptr);
    TMA_NORM_WEIGHT tma_norm_weight(norm_weight_ptr);
    TMA_LINEAR_WEIGHT tma_linear_weight(weight_ptr);
    TMA_OUT tma_out(output_ptr);

    dim3 grid_dim(1, 1, 1);
    dim3 block_dim(256, 1, 1);
    size_t smem_size = 88888;
    hipFuncSetAttribute(reinterpret_cast<const void*>(norm_linear_kernel_hopper_wrapper<T),
                                                           BATCH_SIZE,
                                                           OUTPUT_SIZE,
                                                           REDUCTION_SIZE,
                                                           TMA_INPUT,
                                                           TMA_NORM_WEIGHT,
                                                           TMA_LINEAR_WEIGHT,
                                                           TMA_OUT>,
                         hipFuncAttributeMaxDynamicSharedMemorySize,
                         smem_size);

    norm_linear_kernel_hopper_wrapper<T,
                                      BATCH_SIZE,
                                      OUTPUT_SIZE,
                                      REDUCTION_SIZE,
                                      TMA_INPUT,
                                      TMA_NORM_WEIGHT,
                                      TMA_LINEAR_WEIGHT,
                                      TMA_OUT>
        <<<grid_dim, block_dim, smem_size>>>(
            tma_input, tma_norm_weight, tma_linear_weight, tma_out, eps);
  }

#define DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(                       \
    BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE)                                   \
  case REDUCTION_SIZE:                                                         \
    launch_norm_linear_hopper<bfloat16,                                        \
                              BATCH_SIZE,                                      \
                              OUTPUT_SIZE,                                     \
                              REDUCTION_SIZE>(                                 \
        input_ptr, norm_weight_ptr, weight_ptr, output_ptr, eps);              \
    break;

#define DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE(BATCH_SIZE, OUTPUT_SIZE)    \
  switch (input.size(1)) {                                                     \
    /*DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE, \
    128) DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE,           \
    OUTPUT_SIZE, 256)                                                          \
    DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE,   \
    512) DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(BATCH_SIZE,           \
    OUTPUT_SIZE, 3072)  */                                                     \
    DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE_CASE(                           \
        BATCH_SIZE, OUTPUT_SIZE, 4096)                                         \
    default:                                                                   \
      printf("Unsupported reduction size in test: %zu\n", input.size(1));      \
      break;                                                                   \
  }

#define DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, OUTPUT_SIZE)  \
  case OUTPUT_SIZE:                                                            \
    DISPATCH_NORM_LINEAR_HOPPER_REDUCTION_SIZE(BATCH_SIZE, OUTPUT_SIZE)        \
    break;

#define DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE(BATCH_SIZE)                    \
  switch (output.size(1)) {                                                    \
    DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 16)               \
    DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 32)               \
    DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE_CASE(BATCH_SIZE, 64)               \
    default:                                                                   \
      printf("Unsupported output size in test: %zu\n", output.size(1));        \
      break;                                                                   \
  }

#define DISPATCH_NORM_LINEAR_HOPPER_BATCH_SIZE_CASE(BATCH_SIZE)                \
  case BATCH_SIZE:                                                             \
    DISPATCH_NORM_LINEAR_HOPPER_OUTPUT_SIZE(BATCH_SIZE)                        \
    break;

  void norm_linear_kernel(torch::Tensor input,
                          torch::Tensor norm_weight,
                          torch::Tensor weight,
                          torch::Tensor output,
                          float eps) {

    void *input_ptr = input.data_ptr();
    void *norm_weight_ptr = norm_weight.data_ptr();
    void *weight_ptr = weight.data_ptr();
    void *output_ptr = output.data_ptr();

    switch (input.size(0)) {
      DISPATCH_NORM_LINEAR_HOPPER_BATCH_SIZE_CASE(64)
      default:
        printf("Unsupported output size in test: %zu\n", output.size(0));
        break;
    }

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
  }

  // Multitoken Paged Attention
  template <typename T,
            int NUM_QO_HEADS,
            int NUM_KV_HEADS,
            int KV_CACHE_STRIDE,
            int QKV_STRIDE,
            int O_STRIDE,
            int HEAD_DIM,
            int MAX_SEQ_LEN,
            int PAGE_SIZE,
            typename TMA_Q,
            typename TMA_KV,
            typename TMA_PAGED_KV,
            typename TMA_PAGED_KV_CACHE_TAIL_PAGE,
            typename TMA_OUTPUT,
            int MAX_TOKENS = 8>
  __global__ void multitoken_paged_attention_wrapper_hopper(
      const __grid_constant__ TMA_Q tma_q,
      const __grid_constant__ TMA_KV tma_k,
      const __grid_constant__ TMA_KV tma_v,
      const __grid_constant__ TMA_PAGED_KV tma_paged_k_cache,
      const __grid_constant__ TMA_PAGED_KV tma_paged_v_cache,
      const __grid_constant__ TMA_PAGED_KV_CACHE_TAIL_PAGE
          tma_paged_k_cache_tail_page,
      const __grid_constant__ TMA_PAGED_KV_CACHE_TAIL_PAGE
          tma_paged_v_cache_tail_page,
      const __grid_constant__ TMA_OUTPUT tma_output,
      void *paged_k_cache_ptr,
      void *paged_v_cache_ptr,
      int const *qo_indptr_buffer_ptr,
      int const *paged_kv_indptr_buffer_ptr,
      int const *paged_kv_indices_buffer_ptr,
      int const *paged_kv_last_page_len_buffer_ptr,
      int request_id,
      bool qk_norm,
      bool rope,
      void const *q_norm_weight_ptr,
      void const *k_norm_weight_ptr,
      void const *cos_ptr,
      void const *sin_ptr,
      float q_eps,
      float k_eps) {

    multitoken_paged_attention_hopper_impl<T,
                                           NUM_QO_HEADS,
                                           NUM_KV_HEADS,
                                           KV_CACHE_STRIDE,
                                           QKV_STRIDE,
                                           O_STRIDE,
                                           HEAD_DIM,
                                           MAX_SEQ_LEN,
                                           PAGE_SIZE,
                                           TMA_Q,
                                           TMA_KV,
                                           TMA_PAGED_KV,
                                           TMA_PAGED_KV_CACHE_TAIL_PAGE,
                                           TMA_OUTPUT,
                                           MAX_TOKENS>(
        tma_q,
        tma_k,
        tma_v,
        tma_paged_k_cache,
        tma_paged_v_cache,
        tma_paged_k_cache_tail_page,
        tma_paged_v_cache_tail_page,
        tma_output,
        paged_k_cache_ptr,
        paged_v_cache_ptr,
        qo_indptr_buffer_ptr,
        paged_kv_indptr_buffer_ptr,
        paged_kv_indices_buffer_ptr,
        paged_kv_last_page_len_buffer_ptr,
        request_id,
        qk_norm,
        rope,
        q_norm_weight_ptr,
        k_norm_weight_ptr,
        cos_ptr,
        sin_ptr,
        q_eps,
        k_eps);
  }

  template <typename T,
            int NUM_QO_HEADS,
            int NUM_KV_HEADS,
            int KV_CACHE_STRIDE,
            int QKV_STRIDE,
            int O_STRIDE,
            int HEAD_DIM,
            int MAX_SEQ_LEN,
            int PAGE_SIZE,
            int MAX_TOKENS = 8>
  void launch_multitoken_paged_attention_hopper(
      void *qkv_ptr,
      void *paged_k_cache_ptr,
      void *paged_v_cache_ptr,
      void *output_ptr,
      int const *qo_indptr_buffer_ptr,
      int const *paged_kv_indptr_buffer_ptr,
      int const *paged_kv_indices_buffer_ptr,
      int const *paged_kv_last_page_len_buffer_ptr,
      int request_id,
      bool qk_norm,
      bool rope,
      void const *q_norm_weight_ptr,
      void const *k_norm_weight_ptr,
      void const *cos_ptr,
      void const *sin_ptr,
      float q_eps,
      float k_eps) {
    dim3 grid_dim(1, 1, 1);
    dim3 block_dim(256, 1, 1);
    size_t smem_size = 224 * 1024;

    constexpr int B = 3;
    constexpr int M = 3;
    constexpr int S = 3;
    constexpr int TMA_CP_SIZE = 64;
    constexpr int KV_TILE_SIZE = 64;
    constexpr int prompt_len = 8;
    constexpr int num_tokens = 4;

    constexpr int NUM_PAGES = 100;
    constexpr int TAIL_PAGE_SIZE = prompt_len % PAGE_SIZE;

    using TMA_Q =
        kernel::tma::tma_3d<bfloat16,
                            B,
                            M,
                            S,
                            num_tokens,
                            (NUM_QO_HEADS + 2 * NUM_KV_HEADS),
                            HEAD_DIM,
                            num_tokens,
                            NUM_QO_HEADS,
                            TMA_CP_SIZE,
                            1,
                            (HEAD_DIM + TMA_CP_SIZE - 1) / TMA_CP_SIZE,
                            num_tokens * NUM_QO_HEADS * TMA_CP_SIZE,
                            (NUM_QO_HEADS + 2 * NUM_KV_HEADS) * HEAD_DIM,
                            HEAD_DIM,
                            1,
                            true>;

    using TMA_KV =
        kernel::tma::tma_3d<bfloat16,
                            B,
                            M,
                            S,
                            num_tokens,
                            (NUM_QO_HEADS + 2 * NUM_KV_HEADS),
                            HEAD_DIM,
                            num_tokens,
                            NUM_KV_HEADS,
                            TMA_CP_SIZE,
                            1,
                            (HEAD_DIM + TMA_CP_SIZE - 1) / TMA_CP_SIZE,
                            KV_TILE_SIZE *
                                TMA_CP_SIZE, // skip number of rows between
                                             // current 64 cols and next 64 cols
                            (NUM_QO_HEADS + 2 * NUM_KV_HEADS) * HEAD_DIM,
                            HEAD_DIM,
                            1,
                            true>;

    using TMA_PAGED_KV_CACHE =
        kernel::tma::tma_3d<bfloat16,
                            B,
                            M,
                            S,
                            NUM_PAGES,
                            PAGE_SIZE,
                            HEAD_DIM,
                            1,
                            KV_TILE_SIZE,
                            TMA_CP_SIZE,
                            1,
                            (HEAD_DIM + TMA_CP_SIZE - 1) / TMA_CP_SIZE,
                            KV_TILE_SIZE * TMA_CP_SIZE,
                            PAGE_SIZE * HEAD_DIM,
                            HEAD_DIM,
                            1,
                            true>;

    using TMA_PAGED_KV_CACHE_TAIL_PAGE =
        kernel::tma::tma_3d<bfloat16,
                            B,
                            M,
                            S,
                            NUM_PAGES,
                            PAGE_SIZE,
                            HEAD_DIM,
                            1,
                            TAIL_PAGE_SIZE,
                            TMA_CP_SIZE,
                            1,
                            (HEAD_DIM + TMA_CP_SIZE - 1) / TMA_CP_SIZE,
                            KV_TILE_SIZE * NUM_KV_HEADS * TMA_CP_SIZE,
                            PAGE_SIZE * HEAD_DIM,
                            HEAD_DIM,
                            1,
                            true>;

    using TMA_OUTPUT =
        kernel::tma::tma_2d<bfloat16,
                            3,
                            3,
                            3,
                            num_tokens * NUM_QO_HEADS,
                            HEAD_DIM,
                            num_tokens * NUM_QO_HEADS,
                            TMA_CP_SIZE,
                            1,
                            (HEAD_DIM + TMA_CP_SIZE - 1) / TMA_CP_SIZE,
                            num_tokens * NUM_QO_HEADS * TMA_CP_SIZE,
                            true>;

    bfloat16 *__restrict__ qkv_ptr_bf16 = static_cast<bfloat16 *>(qkv_ptr);

    TMA_Q tma_q(qkv_ptr);
    TMA_KV tma_k(qkv_ptr);
    TMA_KV tma_v(qkv_ptr);
    TMA_PAGED_KV_CACHE tma_paged_k_cache(paged_k_cache_ptr);
    TMA_PAGED_KV_CACHE tma_paged_v_cache(paged_v_cache_ptr);
    TMA_PAGED_KV_CACHE_TAIL_PAGE tma_paged_k_cache_tail_page(paged_k_cache_ptr);
    TMA_PAGED_KV_CACHE_TAIL_PAGE tma_paged_v_cache_tail_page(paged_v_cache_ptr);
    TMA_OUTPUT tma_output(output_ptr);

    hipFuncSetAttribute(reinterpret_cast<const void*>(
        multitoken_paged_attention_wrapper_hopper<T),
                                                  NUM_QO_HEADS,
                                                  NUM_KV_HEADS,
                                                  KV_CACHE_STRIDE,
                                                  QKV_STRIDE,
                                                  O_STRIDE,
                                                  HEAD_DIM,
                                                  MAX_SEQ_LEN,
                                                  PAGE_SIZE,
                                                  TMA_Q,
                                                  TMA_KV,
                                                  TMA_PAGED_KV_CACHE,
                                                  TMA_PAGED_KV_CACHE_TAIL_PAGE,
                                                  TMA_OUTPUT,
                                                  num_tokens>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size);

#ifndef MIRAGE_PROFILE_HOPPER
    multitoken_paged_attention_wrapper_hopper<T,
                                              NUM_QO_HEADS,
                                              NUM_KV_HEADS,
                                              KV_CACHE_STRIDE,
                                              QKV_STRIDE,
                                              O_STRIDE,
                                              HEAD_DIM,
                                              MAX_SEQ_LEN,
                                              PAGE_SIZE,
                                              TMA_Q,
                                              TMA_KV,
                                              TMA_PAGED_KV_CACHE,
                                              TMA_PAGED_KV_CACHE_TAIL_PAGE,
                                              TMA_OUTPUT,
                                              num_tokens>
        <<<grid_dim, block_dim, smem_size>>>(tma_q,
                                             tma_k,
                                             tma_v,
                                             tma_paged_k_cache,
                                             tma_paged_v_cache,
                                             tma_paged_k_cache_tail_page,
                                             tma_paged_v_cache_tail_page,
                                             tma_output,
                                             paged_k_cache_ptr,
                                             paged_v_cache_ptr,
                                             qo_indptr_buffer_ptr,
                                             paged_kv_indptr_buffer_ptr,
                                             paged_kv_indices_buffer_ptr,
                                             paged_kv_last_page_len_buffer_ptr,
                                             request_id,
                                             qk_norm,
                                             rope,
                                             q_norm_weight_ptr,
                                             k_norm_weight_ptr,
                                             cos_ptr,
                                             sin_ptr,
                                             q_eps,
                                             k_eps);
#else

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  constexpr int WARMUP_RUNS = 16;
  constexpr int BENCHMARK_RUNS = 1000;

  printf("=== Multitoken Paged Attention Kernel Performance Profiling ===\n");

  for (int i = 0; i < WARMUP_RUNS; i++) {
    multitoken_paged_attention_wrapper_hopper<T,
                                              NUM_QO_HEADS,
                                              NUM_KV_HEADS,
                                              KV_CACHE_STRIDE,
                                              QKV_STRIDE,
                                              O_STRIDE,
                                              HEAD_DIM,
                                              MAX_SEQ_LEN,
                                              PAGE_SIZE,
                                              TMA_Q,
                                              TMA_KV,
                                              TMA_PAGED_KV_CACHE,
                                              TMA_PAGED_KV_CACHE_TAIL_PAGE,
                                              TMA_OUTPUT,
                                              num_tokens>
        <<<grid_dim, block_dim, smem_size>>>(tma_q,
                                             tma_k,
                                             tma_v,
                                             tma_paged_k_cache,
                                             tma_paged_v_cache,
                                             tma_paged_k_cache_tail_page,
                                             tma_paged_v_cache_tail_page,
                                             tma_output,
                                             paged_k_cache_ptr,
                                             paged_v_cache_ptr,
                                             qo_indptr_buffer_ptr,
                                             paged_kv_indptr_buffer_ptr,
                                             paged_kv_indices_buffer_ptr,
                                             paged_kv_last_page_len_buffer_ptr,
                                             request_id,
                                             qk_norm,
                                             rope,
                                             q_norm_weight_ptr,
                                             k_norm_weight_ptr,
                                             cos_ptr,
                                             sin_ptr,
                                             q_eps,
                                             k_eps);
  }
  hipDeviceSynchronize();

  printf("Running %d benchmark iterations...\n", BENCHMARK_RUNS);

  float *iteration_times = new float[BENCHMARK_RUNS];
  float total_time_ms = 0.0f;

  for (int i = 0; i < BENCHMARK_RUNS; i++) {
    hipEventRecord(start);
    multitoken_paged_attention_wrapper_hopper<T,
                                              NUM_QO_HEADS,
                                              NUM_KV_HEADS,
                                              KV_CACHE_STRIDE,
                                              QKV_STRIDE,
                                              O_STRIDE,
                                              HEAD_DIM,
                                              MAX_SEQ_LEN,
                                              PAGE_SIZE,
                                              TMA_Q,
                                              TMA_KV,
                                              TMA_PAGED_KV_CACHE,
                                              TMA_PAGED_KV_CACHE_TAIL_PAGE,
                                              TMA_OUTPUT,
                                              num_tokens>
        <<<grid_dim, block_dim, smem_size>>>(tma_q,
                                             tma_k,
                                             tma_v,
                                             tma_paged_k_cache,
                                             tma_paged_v_cache,
                                             tma_paged_k_cache_tail_page,
                                             tma_paged_v_cache_tail_page,
                                             tma_output,
                                             paged_k_cache_ptr,
                                             paged_v_cache_ptr,
                                             qo_indptr_buffer_ptr,
                                             paged_kv_indptr_buffer_ptr,
                                             paged_kv_indices_buffer_ptr,
                                             paged_kv_last_page_len_buffer_ptr,
                                             request_id,
                                             qk_norm,
                                             rope,
                                             q_norm_weight_ptr,
                                             k_norm_weight_ptr,
                                             cos_ptr,
                                             sin_ptr,
                                             q_eps,
                                             k_eps);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float iteration_time_ms;
    hipEventElapsedTime(&iteration_time_ms, start, stop);

    iteration_times[i] = iteration_time_ms;
    total_time_ms += iteration_time_ms;
  }

  float avg_time_ms = total_time_ms / BENCHMARK_RUNS;

  printf("\n=== Multitoken Paged Attention Performance Results ===\n");
  printf("Configuration:\n");
  printf("  NUM_QO_HEADS=%d, NUM_KV_HEADS=%d, HEAD_DIM=%d\n",
         NUM_QO_HEADS,
         NUM_KV_HEADS,
         HEAD_DIM);
  printf("  MAX_SEQ_LEN=%d, PAGE_SIZE=%d, MAX_TOKENS=%d\n",
         MAX_SEQ_LEN,
         PAGE_SIZE,
         MAX_TOKENS);
  printf("  Average: %.3f ms\n", avg_time_ms);

  printf("===============================\n");

  delete[] iteration_times;
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif
  }

  void multitoken_paged_attention_hopper(
      torch::Tensor qkv,
      torch::Tensor paged_k_cache,
      torch::Tensor paged_v_cache,
      torch::Tensor output,
      torch::Tensor qo_indptr_buffer,
      torch::Tensor paged_kv_indptr_buffer,
      torch::Tensor paged_kv_indices_buffer,
      torch::Tensor paged_kv_last_page_len_buffer,
      int request_id,
      bool qk_norm,
      bool rope,
      torch::optional<torch::Tensor> q_norm_weight = torch::nullopt,
      torch::optional<torch::Tensor> k_norm_weight = torch::nullopt,
      torch::optional<torch::Tensor> cos = torch::nullopt,
      torch::optional<torch::Tensor> sin = torch::nullopt,
      float q_eps = 0.0f,
      float k_eps = 0.0f) {
    void *qkv_ptr = qkv.data_ptr();
    void *paged_k_cache_ptr = paged_k_cache.data_ptr();
    void *paged_v_cache_ptr = paged_v_cache.data_ptr();
    void *output_ptr = output.data_ptr();
    int const *qo_indptr_buffer_ptr = qo_indptr_buffer.data_ptr<int>();
    int const *paged_kv_indptr_buffer_ptr =
        paged_kv_indptr_buffer.data_ptr<int>();
    int const *paged_kv_indices_buffer_ptr =
        paged_kv_indices_buffer.data_ptr<int>();
    int const *paged_kv_last_page_len_buffer_ptr =
        paged_kv_last_page_len_buffer.data_ptr<int>();

    void const *q_norm_weight_ptr =
        qk_norm ? q_norm_weight->data_ptr() : nullptr;
    void const *k_norm_weight_ptr =
        qk_norm ? k_norm_weight->data_ptr() : nullptr;
    void const *cos_ptr = rope ? cos->data_ptr() : nullptr;
    void const *sin_ptr = rope ? sin->data_ptr() : nullptr;
    int const qo_heads = 4;
    int const kv_heads = 1;
    int const head_dim = 128;
    int const qkv_stride = (qo_heads + 2 * kv_heads) * head_dim;
    assert(qkv_stride == qkv.stride(0));
    int const kv_stride = head_dim * kv_heads;
    assert(kv_stride == paged_k_cache.stride(1));
    int const o_stride = head_dim * qo_heads;
    int const page_size = 4096;
    int const max_seq_len = 512;

    launch_multitoken_paged_attention_hopper<bfloat16,
                                             qo_heads,
                                             kv_heads,
                                             kv_stride,
                                             qkv_stride,
                                             o_stride,
                                             head_dim,
                                             max_seq_len,
                                             page_size>(
        qkv_ptr,
        paged_k_cache_ptr,
        paged_v_cache_ptr,
        output_ptr,
        qo_indptr_buffer_ptr,
        paged_kv_indptr_buffer_ptr,
        paged_kv_indices_buffer_ptr,
        paged_kv_last_page_len_buffer_ptr,
        request_id,
        qk_norm,
        rope,
        q_norm_weight_ptr,
        k_norm_weight_ptr,
        cos_ptr,
        sin_ptr,
        q_eps,
        k_eps);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
  }

  PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("linear", &linear_kernel, "Linear kernel");
    m.def("norm_linear", &norm_linear_kernel, "NormLinear kernel");
    m.def("multitoken_paged_attention",
          &multitoken_paged_attention_hopper,
          "Multitoken paged attention for Grace Hopper GPU");
  }
