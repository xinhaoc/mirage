#include "hip/hip_runtime.h"
/* Copyright 2025 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "blackwell/task_header.cuh"
#include "hopper/tma_2d.cuh"
#include "runtime_header.h"
#include "tma.cuh"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <cstdio>
#include <iostream>

// Cutlass includes
#include <cutlass/arch/barrier.h>
#include <cutlass/cluster_launch.hpp>
#include <cutlass/half.h> // F16 data type
#include <cutlass/util/print_error.hpp>

// CuTe includes
#include <cute/algorithm/cooperative_copy.hpp> // Auto vectorized copy operation
#include <cute/arch/cluster_sm90.hpp> // CuTe functions for querying the details of cluster launched
#include <cute/arch/tmem_allocator_sm100.hpp> // TMEM allocator for SM100
#include <cute/numeric/integral_constant.hpp> // Compile time in constants such as _1, _256 etc.
#include <cute/pointer_flagged.hpp>
#include <cute/tensor.hpp> // CuTe tensor implementation

using bfloat16 = cute::bfloat16_t;

// topk_softmax_sm100

template <typename T, int EXPERTS, int BYTES_PER_LDG>
__global__ __launch_bounds__(256) void topk_softmax_kernel(
    T const *__restrict__ gating_output,
    float *__restrict__ topk_weights,
    int *__restrict__ mpk_routing_indices, // [EXPERTS, num_rows] expert-major
    int *__restrict__ mpk_expert_mask,     // [EXPERTS]
    int num_rows,
    int k,
    bool renormalize) {
  using C = kernel::detail::TopkConstants<T, EXPERTS, BYTES_PER_LDG>;
  static constexpr int VPT = C::VPT;
  static constexpr int WARPS_PER_TB = 8; // 256 threads
  kernel::topk_softmax_task_impl<T, VPT, EXPERTS, WARPS_PER_TB, BYTES_PER_LDG>(
      gating_output,
      /*finished*/ nullptr,
      topk_weights,
      num_rows,
      k,
      mpk_routing_indices,
      mpk_expert_mask,
      /*start_expert=*/0,
      /*end_expert=*/EXPERTS,
      renormalize);
}

// New: expose a direct fused TopK softmax without GEMM
void topk_softmax_sm100_kernel(torch::Tensor gating_output,
                               torch::Tensor topk_weights,
                               torch::Tensor mpk_routing_indices,
                               torch::Tensor mpk_expert_mask) {

  int const BATCH_SIZE = static_cast<int>(gating_output.size(0));
  int const OUTPUT_SIZE = static_cast<int>(gating_output.size(1));
  int const NUM_TOPK = static_cast<int>(topk_weights.size(1));

  assert(topk_weights.size(0) == BATCH_SIZE &&
         topk_weights.size(1) == NUM_TOPK);
  assert(mpk_routing_indices.size(0) == OUTPUT_SIZE &&
         mpk_routing_indices.size(1) == BATCH_SIZE);
  assert(mpk_expert_mask.size(0) == OUTPUT_SIZE);

  // launch grid using 256-thread blocks
  auto launch = [&](auto experts_ct) {
    constexpr int EXP = decltype(experts_ct)::value;
    using T = bfloat16;
    dim3 grid_dim(1, 1, 1);
    dim3 block_dim(256, 1, 1);
    topk_softmax_kernel<T,
                        EXP,
                        ((sizeof(T) * EXP) < 16 ? (sizeof(T) * EXP) : 16)>
        <<<grid_dim, block_dim, 0>>>(
            static_cast<const T *>(gating_output.data_ptr()),
            topk_weights.data_ptr<float>(),
            mpk_routing_indices.data_ptr<int>(),
            mpk_expert_mask.data_ptr<int>(),
            BATCH_SIZE,
            NUM_TOPK,
            /*renormalize=*/true);
  };

  switch (OUTPUT_SIZE) {
    case 1:
      launch(std::integral_constant<int, 1>{});
      break;
    case 2:
      launch(std::integral_constant<int, 2>{});
      break;
    case 4:
      launch(std::integral_constant<int, 4>{});
      break;
    case 8:
      launch(std::integral_constant<int, 8>{});
      break;
    case 16:
      launch(std::integral_constant<int, 16>{});
      break;
    case 32:
      launch(std::integral_constant<int, 32>{});
      break;
    case 64:
      launch(std::integral_constant<int, 64>{});
      break;
    case 128:
      launch(std::integral_constant<int, 128>{});
      break;
    case 256:
      launch(std::integral_constant<int, 256>{});
      break;
    default:
      printf("Unsupported num_experts=%d (must be power-of-two <= 256)\n",
             OUTPUT_SIZE);
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// moe_linear_sm100

template <typename T,
          int BATCH_SIZE,
          int OUTPUT_SIZE,
          int REDUCTION_SIZE,
          int NUM_EXPERTS,
          int NUM_TOPK,
          int EXPERT_OFFSET,
          int EXPERT_STRIDE,
          bool W13_LINEAR,
          class InputTensor,
          class BiasTensor,
          class IndicesTensor,
          class MaskTensor,
          class OutputTensor,
          int MMA_M,
          int MMA_N,
          bool NoBias,
          int NUM_AB_STAGE = 8,
          int NUM_ACC_STAGE = 2,
          int NUM_C_STAGE = 4>
__global__ __launch_bounds__(256, 1) void moe_linear_sm100_wrapper(
    void *tma_w_desc_ptr,
    InputTensor mInput,
    BiasTensor mBias,
    IndicesTensor mRoutingIndices,
    MaskTensor mMask,
    OutputTensor mOutput) {

  constexpr int B = 3;
  constexpr int M = 3;
  constexpr int S = 3;

  constexpr int TMA_CP_ASYNC_SIZE =
      64; // note that if swizzle 128 is used, 64 is maximal cp size
  constexpr int TILE_SIZE =
      64; // we should modify this param if we want larger tile size
  constexpr int TMA_CP_ASYNC_REPEAT_COL =
      (TILE_SIZE + TMA_CP_ASYNC_SIZE - 1) / TMA_CP_ASYNC_SIZE;

  using TMA_A =
      kernel::tma::tma_2d<bfloat16,
                          B,
                          M,
                          S,
                          NUM_EXPERTS * OUTPUT_SIZE, /*GMEM_ROW_*/
                          REDUCTION_SIZE,            /*GMEM_COL_*/
                          MMA_M,                     /*SMEM_ROW_*/
                          TMA_CP_ASYNC_SIZE,         /*SMEM_COL_*/
                          REDUCTION_SIZE,            /*GMEM_STRIDE_ROW_*/
                          1,                         /*GMEM_STRIDE_COL_*/
                          1,                         /*SMEM_REPEAT_ROW_*/
                          TMA_CP_ASYNC_REPEAT_COL,   /*SMEM_REPEAT_COL_*/
                          MMA_M * TMA_CP_ASYNC_SIZE, /*SMEM_STRIDE_*/
                          true>;

  TMA_A tma_a(static_cast<CUtensorMap *>(tma_w_desc_ptr));

  kernel::moe_linear_sm100_task_impl<T,
                                     TMA_A,
                                     InputTensor,
                                     BiasTensor,
                                     IndicesTensor,
                                     MaskTensor,
                                     OutputTensor,
                                     MMA_M,
                                     MMA_N,
                                     BATCH_SIZE,
                                     OUTPUT_SIZE,
                                     REDUCTION_SIZE,
                                     NUM_EXPERTS,
                                     NUM_TOPK,
                                     EXPERT_OFFSET,
                                     EXPERT_STRIDE,
                                     W13_LINEAR,
                                     NoBias,
                                     NUM_AB_STAGE,
                                     NUM_ACC_STAGE,
                                     NUM_C_STAGE>(
      tma_a, mInput, mBias, mRoutingIndices, mMask, mOutput);
}

template <typename T,
          int BATCH_SIZE,
          int OUTPUT_SIZE,
          int REDUCTION_SIZE,
          int NUM_EXPERTS,
          int NUM_TOPK,
          int EXPERT_OFFSET,
          int EXPERT_STRIDE,
          bool W13_LINEAR = true>
void launch_moe_linear_sm100(void *input_ptr,
                             void *weight_ptr,
                             void *mpk_routing_indices_ptr,
                             void *mpk_expert_mask_ptr,
                             void *output_ptr,
                             void *residual_ptr = nullptr) {

  constexpr int B = 3;
  constexpr int M = 3;
  constexpr int S = 3;

  constexpr int MMA_M = 128;
  constexpr int MMA_N = 16;

  constexpr int TMA_CP_ASYNC_SIZE =
      64; // note that if swizzle 128 is used, 64 is maximal cp size
  constexpr int TILE_SIZE =
      64; // we should modify this param if we want larger tile size

  CUtensorMap host_w_desc;
  CUtensorMap *desc_w_ptr;

  // TMA_WEIGHT, for input we do cp_async in the kernel
  uint64_t w_gmem_shape[2] = {static_cast<uint64_t>(NUM_EXPERTS * OUTPUT_SIZE),
                              static_cast<uint64_t>(REDUCTION_SIZE)};
  uint64_t w_gmem_stride[2] = {1, static_cast<uint64_t>(REDUCTION_SIZE)};
  uint32_t w_smem_shape[2] = {static_cast<uint32_t>(MMA_M),
                              static_cast<uint32_t>(TMA_CP_ASYNC_SIZE)};
  size_t w_smem_repeat_col =
      (TILE_SIZE + TMA_CP_ASYNC_SIZE - 1) / TMA_CP_ASYNC_SIZE;
  mirage::runtime::fill_tma_desc<bfloat16, B, M, S, 2>(
      &host_w_desc,
      static_cast<bfloat16 *>(weight_ptr),
      w_gmem_shape,
      w_gmem_stride,
      w_smem_shape,
      1,
      w_smem_repeat_col);

  hipMalloc(&desc_w_ptr, sizeof(CUtensorMap));
  hipMemcpy(
      desc_w_ptr, &host_w_desc, sizeof(CUtensorMap), hipMemcpyHostToDevice);

  void *tma_desc_weight;
  tma_desc_weight = desc_w_ptr;

  // Residual
  cute::Layout layout_bias = cute::make_layout(
      cute::make_shape(BATCH_SIZE, OUTPUT_SIZE, NUM_EXPERTS),
      cute::make_stride(OUTPUT_SIZE, cute::Int<1>{}, BATCH_SIZE * OUTPUT_SIZE));
  cute::Tensor mBias = cute::make_tensor(
      cute::make_gmem_ptr(static_cast<T *>(residual_ptr)), layout_bias);

  // Topk_indices
  cute::Layout layout_routing_indices =
      cute::make_layout(cute::make_shape(NUM_EXPERTS, BATCH_SIZE),
                        cute::make_stride(BATCH_SIZE, cute::Int<1>{}));
  cute::Tensor mRoutingIndices = cute::make_tensor(
      cute::make_gmem_ptr(static_cast<int32_t *>(mpk_routing_indices_ptr)),
      layout_routing_indices);

  // Topk_weights
  cute::Layout layout_expert_mask = cute::make_layout(
      cute::make_shape(NUM_EXPERTS), cute::make_stride(cute::Int<1>{}));
  cute::Tensor mMask = cute::make_tensor(
      cute::make_gmem_ptr(static_cast<int32_t *>(mpk_expert_mask_ptr)),
      layout_expert_mask);

  // Output
  cute::Layout layout_output = cute::make_layout(
      cute::make_shape(BATCH_SIZE, NUM_TOPK, OUTPUT_SIZE),
      cute::make_stride(NUM_TOPK * OUTPUT_SIZE, OUTPUT_SIZE, cute::Int<1>{}));
  cute::Tensor mOutput = cute::make_tensor(
      cute::make_gmem_ptr(static_cast<T *>(output_ptr)), layout_output);

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(256, 1, 1);
  dim3 cluster_dim(1, 1, 1);
  int smemBytes = 224 * 1024;

  // Input
  if constexpr (W13_LINEAR) {
    cute::Layout layout_input =
        cute::make_layout(cute::make_shape(BATCH_SIZE, REDUCTION_SIZE),
                          cute::make_stride(REDUCTION_SIZE, cute::Int<1>{}));
    cute::Tensor mInput = cute::make_tensor(
        cute::make_gmem_ptr(static_cast<T *>(input_ptr)), layout_input);
    if (residual_ptr != nullptr) {
      auto *kernel_ptr = &moe_linear_sm100_wrapper<T,
                                                   BATCH_SIZE,
                                                   OUTPUT_SIZE,
                                                   REDUCTION_SIZE,
                                                   NUM_EXPERTS,
                                                   NUM_TOPK,
                                                   EXPERT_OFFSET,
                                                   EXPERT_STRIDE,
                                                   W13_LINEAR,
                                                   decltype(mInput),
                                                   decltype(mBias),
                                                   decltype(mRoutingIndices),
                                                   decltype(mMask),
                                                   decltype(mOutput),
                                                   MMA_M,
                                                   MMA_N,
                                                   false>;
      CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
      cutlass::ClusterLaunchParams params = {
          grid_dim, block_dim, cluster_dim, smemBytes};
      cutlass::Status status =
          cutlass::launch_kernel_on_cluster(params,
                                            (void const *)kernel_ptr,
                                            tma_desc_weight,
                                            mInput,
                                            mBias,
                                            mRoutingIndices,
                                            mMask,
                                            mOutput);
      CUTE_CHECK_LAST();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Error: Failed at kernel Launch" << std::endl;
      }
    } else {
      auto *kernel_ptr = &moe_linear_sm100_wrapper<T,
                                                   BATCH_SIZE,
                                                   OUTPUT_SIZE,
                                                   REDUCTION_SIZE,
                                                   NUM_EXPERTS,
                                                   NUM_TOPK,
                                                   EXPERT_OFFSET,
                                                   EXPERT_STRIDE,
                                                   W13_LINEAR,
                                                   decltype(mInput),
                                                   decltype(mBias),
                                                   decltype(mRoutingIndices),
                                                   decltype(mMask),
                                                   decltype(mOutput),
                                                   MMA_M,
                                                   MMA_N,
                                                   true>;
      CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
      cutlass::ClusterLaunchParams params = {
          grid_dim, block_dim, cluster_dim, smemBytes};
      cutlass::Status status =
          cutlass::launch_kernel_on_cluster(params,
                                            (void const *)kernel_ptr,
                                            tma_desc_weight,
                                            mInput,
                                            mBias,
                                            mRoutingIndices,
                                            mMask,
                                            mOutput);
      CUTE_CHECK_LAST();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Error: Failed at kernel Launch" << std::endl;
      }
    }
  } else {
    cute::Layout layout_input = cute::make_layout(
        cute::make_shape(BATCH_SIZE, REDUCTION_SIZE, NUM_TOPK),
        cute::make_stride(
            REDUCTION_SIZE * NUM_TOPK, cute::Int<1>{}, REDUCTION_SIZE));
    cute::Tensor mInput = cute::make_tensor(
        cute::make_gmem_ptr(static_cast<T *>(input_ptr)), layout_input);
    if (residual_ptr != nullptr) {
      auto *kernel_ptr = &moe_linear_sm100_wrapper<T,
                                                   BATCH_SIZE,
                                                   OUTPUT_SIZE,
                                                   REDUCTION_SIZE,
                                                   NUM_EXPERTS,
                                                   NUM_TOPK,
                                                   EXPERT_OFFSET,
                                                   EXPERT_STRIDE,
                                                   W13_LINEAR,
                                                   decltype(mInput),
                                                   decltype(mBias),
                                                   decltype(mRoutingIndices),
                                                   decltype(mMask),
                                                   decltype(mOutput),
                                                   MMA_M,
                                                   MMA_N,
                                                   false>;
      CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
      cutlass::ClusterLaunchParams params = {
          grid_dim, block_dim, cluster_dim, smemBytes};
      cutlass::Status status =
          cutlass::launch_kernel_on_cluster(params,
                                            (void const *)kernel_ptr,
                                            tma_desc_weight,
                                            mInput,
                                            mBias,
                                            mRoutingIndices,
                                            mMask,
                                            mOutput);
      CUTE_CHECK_LAST();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Error: Failed at kernel Launch" << std::endl;
      }
    } else {
      auto *kernel_ptr = &moe_linear_sm100_wrapper<T,
                                                   BATCH_SIZE,
                                                   OUTPUT_SIZE,
                                                   REDUCTION_SIZE,
                                                   NUM_EXPERTS,
                                                   NUM_TOPK,
                                                   EXPERT_OFFSET,
                                                   EXPERT_STRIDE,
                                                   W13_LINEAR,
                                                   decltype(mInput),
                                                   decltype(mBias),
                                                   decltype(mRoutingIndices),
                                                   decltype(mMask),
                                                   decltype(mOutput),
                                                   MMA_M,
                                                   MMA_N,
                                                   true>;
      CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
      cutlass::ClusterLaunchParams params = {
          grid_dim, block_dim, cluster_dim, smemBytes};
      cutlass::Status status =
          cutlass::launch_kernel_on_cluster(params,
                                            (void const *)kernel_ptr,
                                            tma_desc_weight,
                                            mInput,
                                            mBias,
                                            mRoutingIndices,
                                            mMask,
                                            mOutput);
      CUTE_CHECK_LAST();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Error: Failed at kernel Launch" << std::endl;
      }
    }
  }
}

void moe_w13_linear_sm100_kernel(torch::Tensor input,
                                 torch::Tensor weight,
                                 c10::optional<at::Tensor> residual,
                                 torch::Tensor mpk_routing_indices,
                                 torch::Tensor mpk_expert_mask,
                                 torch::Tensor output) {

  void *input_ptr = input.data_ptr();
  void *weight_ptr = weight.data_ptr();
  bool has_residual = residual.has_value();
  void *residual_ptr = has_residual ? residual->data_ptr() : nullptr;
  void *mpk_routing_indices_ptr = mpk_routing_indices.data_ptr();
  void *mpk_expert_mask_ptr = mpk_expert_mask.data_ptr();
  void *output_ptr = output.data_ptr();

  // const int BATCH_SIZE = input.size(0);
  // const int OUTPUT_SIZE = output.size(1);
  // const int REDUCTION_SIZE = weight.size(1);

  constexpr int BATCH_SIZE = 8;
  constexpr int OUTPUT_SIZE = 128;
  constexpr int REDUCTION_SIZE = 2048;
  constexpr int NUM_EXPERTS = 128;
  constexpr int NUM_TOPK = 8;
  constexpr int EXPERT_OFFSET = 0;
  constexpr int EXPERT_STRIDE = 12;

  assert(input.size(1) == REDUCTION_SIZE);
  assert(weight.size(0) == NUM_EXPERTS && weight.size(1) == OUTPUT_SIZE &&
         weight.size(2) == REDUCTION_SIZE);
  assert(mpk_routing_indices.size(0) == NUM_EXPERTS &&
         mpk_routing_indices.size(1) == BATCH_SIZE);
  assert(mpk_expert_mask.size(0) == NUM_EXPERTS);
  assert(!has_residual);

  launch_moe_linear_sm100<bfloat16,
                          BATCH_SIZE,
                          OUTPUT_SIZE,
                          REDUCTION_SIZE,
                          NUM_EXPERTS,
                          NUM_TOPK,
                          EXPERT_OFFSET,
                          EXPERT_STRIDE,
                          true>(input_ptr,
                                weight_ptr,
                                mpk_routing_indices_ptr,
                                mpk_expert_mask_ptr,
                                output_ptr,
                                residual_ptr);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

void moe_w2_linear_sm100_kernel(torch::Tensor input,
                                torch::Tensor weight,
                                c10::optional<at::Tensor> residual,
                                torch::Tensor mpk_routing_indices,
                                torch::Tensor mpk_expert_mask,
                                torch::Tensor output) {

  void *input_ptr = input.data_ptr();
  void *weight_ptr = weight.data_ptr();
  bool has_residual = residual.has_value();
  void *residual_ptr = has_residual ? residual->data_ptr() : nullptr;
  void *mpk_routing_indices_ptr = mpk_routing_indices.data_ptr();
  void *mpk_expert_mask_ptr = mpk_expert_mask.data_ptr();
  void *output_ptr = output.data_ptr();

  // const int BATCH_SIZE = input.size(0);
  // const int OUTPUT_SIZE = output.size(1);
  // const int REDUCTION_SIZE = weight.size(1);

  constexpr int BATCH_SIZE = 8;
  constexpr int OUTPUT_SIZE = 128;
  constexpr int REDUCTION_SIZE = 2048;
  constexpr int NUM_EXPERTS = 128;
  constexpr int NUM_TOPK = 8;
  constexpr int EXPERT_OFFSET = 0;
  constexpr int EXPERT_STRIDE = 12;

  assert(input.size(0) == BATCH_SIZE && input.size(1) == NUM_TOPK &&
         input.size(2) == REDUCTION_SIZE);
  assert(weight.size(0) == NUM_EXPERTS && weight.size(1) == OUTPUT_SIZE &&
         weight.size(2) == REDUCTION_SIZE);
  assert(mpk_routing_indices.size(0) == NUM_EXPERTS &&
         mpk_routing_indices.size(1) == BATCH_SIZE);
  assert(mpk_expert_mask.size(0) == NUM_EXPERTS);
  assert(!has_residual);

  launch_moe_linear_sm100<bfloat16,
                          BATCH_SIZE,
                          OUTPUT_SIZE,
                          REDUCTION_SIZE,
                          NUM_EXPERTS,
                          NUM_TOPK,
                          EXPERT_OFFSET,
                          EXPERT_STRIDE,
                          false>(input_ptr,
                                 weight_ptr,
                                 mpk_routing_indices_ptr,
                                 mpk_expert_mask_ptr,
                                 output_ptr,
                                 residual_ptr);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// mul_sum_add_sm100

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int NUM_TOPK>
__global__ __launch_bounds__(256) void mul_sum_add_sm100_wrapper(
    void const *input_ptr,
    void const *residual_ptr,
    void const *weight_ptr,
    void *output_ptr) {
  kernel::mul_sum_add_sm100_task_impl<T, BATCH_SIZE, OUTPUT_SIZE, NUM_TOPK>(
      input_ptr, residual_ptr, weight_ptr, output_ptr);
}

void mul_sum_add_sm100_kernel(torch::Tensor input,
                              torch::Tensor residual,
                              torch::Tensor weight,
                              torch::Tensor output) {

  using T = bfloat16;

  void *input_ptr = input.data_ptr();
  void *residual_ptr = residual.data_ptr();
  void *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  constexpr int BATCH_SIZE = 1;
  constexpr int OUTPUT_SIZE = 256;
  constexpr int NUM_TOPK = 8;

  assert(input.size(0) == BATCH_SIZE && input.size(1) == NUM_TOPK &&
         input.size(2) == OUTPUT_SIZE);
  assert(residual.size(0) == BATCH_SIZE && residual.size(1) == OUTPUT_SIZE);
  assert(weight.size(0) == BATCH_SIZE && weight.size(1) == NUM_TOPK);
  assert(output.size(0) == BATCH_SIZE && output.size(1) == OUTPUT_SIZE);

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(256, 1, 1);
  dim3 cluster_dim(1, 1, 1);
  int smemBytes = 224 * 1024;

  auto *kernel_ptr =
      &mul_sum_add_sm100_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, NUM_TOPK>;
  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
      kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
  cutlass::ClusterLaunchParams params = {
      grid_dim, block_dim, cluster_dim, smemBytes};
  cutlass::Status status =
      cutlass::launch_kernel_on_cluster(params,
                                        (void const *)kernel_ptr,
                                        input_ptr,
                                        residual_ptr,
                                        weight_ptr,
                                        output_ptr);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

// silu_mul

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int NUM_TOPK>
__global__ __launch_bounds__(256) void silu_mul_wrapper(void const *input_ptr,
                                                        void *output_ptr) {
  constexpr int INTER_SIZE = OUTPUT_SIZE * 2;
  kernel::
      silu_mul_task_impl<T, BATCH_SIZE, OUTPUT_SIZE, INTER_SIZE, OUTPUT_SIZE>(
          input_ptr, output_ptr, BATCH_SIZE * NUM_TOPK);
}

void silu_mul_kernel(torch::Tensor input, torch::Tensor output) {

  using T = bfloat16;

  void *input_ptr = input.data_ptr();
  void *output_ptr = output.data_ptr();

  constexpr int BATCH_SIZE = 1;
  constexpr int OUTPUT_SIZE = 768;
  constexpr int NUM_TOPK = 1;

  assert(input.size(0) == BATCH_SIZE && input.size(1) == NUM_TOPK &&
         input.size(2) == OUTPUT_SIZE * 2);
  assert(output.size(0) == BATCH_SIZE && output.size(1) == NUM_TOPK &&
         output.size(2) == OUTPUT_SIZE);

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(256, 1, 1);
  dim3 cluster_dim(1, 1, 1);
  int smemBytes = 224 * 1024;

  auto *kernel_ptr = &silu_mul_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, NUM_TOPK>;
  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
      kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smemBytes));
  cutlass::ClusterLaunchParams params = {
      grid_dim, block_dim, cluster_dim, smemBytes};
  cutlass::Status status = cutlass::launch_kernel_on_cluster(
      params, (void const *)kernel_ptr, input_ptr, output_ptr);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("topk_softmax_sm100",
        &topk_softmax_sm100_kernel,
        "TopK Softmax fused SM100");
  m.def("moe_w13_linear_sm100",
        &moe_w13_linear_sm100_kernel,
        "MoE W13 Linear kernel SM100");
  m.def("moe_w2_linear_sm100",
        &moe_w2_linear_sm100_kernel,
        "MoE W2 Linear kernel SM100");
  m.def("mul_sum_add_sm100",
        &mul_sum_add_sm100_kernel,
        "Mul Sum Add kernel SM100");
  m.def("silu_mul", &silu_mul_kernel, "SiLU Mul kernel SM100");
}