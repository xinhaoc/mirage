#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static constexpr int SINGLE_KERNEL_THREADS = 128;
static constexpr size_t SM_COUNT = 96;
static constexpr size_t ARRAY_SIZE = 1024 * 1024; // 1M elements

// Device helper function that adds two numbers
__device__ __noinline__ float add_two_numbers(float a, float b) {
    return a + b;
}

// Kernel that takes 2 input pointers and 1 output pointer
// Delegates all work to the device helper function
// __global__ void main_kernel(float *input1, float *input2, float *output) {
//     // Calculate global thread index
//     size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     size_t stride = gridDim.x * blockDim.x;
    
//     // Grid-stride loop to handle all elements
//     for (size_t i = idx; i < ARRAY_SIZE; i += stride) {
//         // Delegate the addition to the device helper function
//         output[i] = add_two_numbers(input1[i], input2[i]);
//     }
// }

#define CUDA_CHECK(err) do { if (err != hipSuccess) { printf("CUDA error: %d\n", err); return 1; } } while (0)

int main() {
    hipDevice_t cu_device;
    hipCtx_t cu_context;
    hipModule_t mod;
    hipFunction_t main_kernel;
    hipInit(0);
    hipDeviceGet(&cu_device, 0);
    hipCtxCreate(&cu_context, 0, cu_device);


    auto result = hipModuleLoad(&mod, "test_param.ptx");
    if (result != hipSuccess) {
        printf("Error loading module: %d\n", result);
        return 1;
    }
    auto result2 = hipModuleGetFunction(&main_kernel, mod, "_Z11main_kernelPfS_S_");
    if (result2 != hipSuccess) {
        printf("Error getting function: %d\n", result2);
        return 1;
    }



    // int device;
    // hipGetDevice(&device);
    // int sm_count;
    // hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);
    // printf("Running kernel with %d SMs\n", sm_count);
    // printf("Array size: %zu elements\n", ARRAY_SIZE);

    // Allocate device memory for inputs and output
    hipDeviceptr_t d_input1 = NULL;
    hipDeviceptr_t d_input2 = NULL;
    hipDeviceptr_t d_output = NULL;

    size_t array_bytes = ARRAY_SIZE * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_input1, array_bytes));
    CUDA_CHECK(hipMalloc(&d_input2, array_bytes));
    CUDA_CHECK(hipMalloc(&d_output, array_bytes));

    // Allocate and initialize host buffers
    float *h_input1 = (float*)malloc(array_bytes);
    float *h_input2 = (float*)malloc(array_bytes);
    float *h_output = (float*)malloc(array_bytes);

    // Fill input arrays with test values
    for (size_t i = 0; i < ARRAY_SIZE; ++i) {
        h_input1[i] = 1.0f;
        h_input2[i] = 2.0f;
    }

    // Copy inputs to device
    // hipMemcpy(d_input1, h_input1, array_bytes, hipMemcpyHostToDevice);
    // hipMemcpy(d_input2, h_input2, array_bytes, hipMemcpyHostToDevice);
    CUDA_CHECK(hipMemcpyHtoD(d_input1, h_input1, array_bytes));
    CUDA_CHECK(hipMemcpyHtoD(d_input2, h_input2, array_bytes));

    hipStream_t stream;
    auto stream_result = hipStreamCreateWithFlags(&stream, 0);
    if (stream_result != hipSuccess) {
        printf("Error creating stream: %d\n", stream_result);
        return 1;
    }


    // Launch kernel with same grid and block dimensions as original
    // Grid: (sm_count, 1, 1) = (96, 1, 1)
    // Block: (SINGLE_KERNEL_THREADS, 1, 1) = (128, 1, 1)
    // main_kernel<<<dim3(sm_count, 1, 1), 
    //                dim3(SINGLE_KERNEL_THREADS, 1, 1)>>>(
    //     d_input1, d_input2, d_output
    // );
    void *params[3] = {&d_input1, &d_input2, &d_output};
    CUDA_CHECK(cuFuncSetAttribute(main_kernel, HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES, 160 * 1024));
    auto launch_result = hipModuleLaunchKernel(main_kernel, 1, 1, 1, 1, 1, 1, 160 * 1024, NULL, (void**) params, NULL);
    if (launch_result != hipSuccess) {
        printf("Error launching kernel: %d\n", launch_result);
        return 1;
    }


    // Check for launch errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy result back to host
    // hipMemcpy(h_output, d_output, array_bytes, hipMemcpyDeviceToHost);
    CUDA_CHECK(hipMemcpyDtoH(h_output, d_output, array_bytes));

    // Verify results (1.0 + 2.0 = 3.0)
    bool success = true;
    for (size_t i = 0; i < ARRAY_SIZE; ++i) {
        if (h_output[i] != 3.0f) {
            printf("Error: h_output[%zu] = %f (expected 3.0)\n", i, h_output[i]);
            success = false;
            break;
        }
    }

    if (success) {
        printf("Success! All results are correct.\n");
    }

    // Cleanup
    free(h_input1);
    free(h_input2);
    free(h_output);
    CUDA_CHECK(hipFree(d_input1));
    CUDA_CHECK(hipFree(d_input2));
    CUDA_CHECK(hipFree(d_output));
    // hipFree(d_input2);
    // hipFree(d_output);

    return success ? 0 : 1;
}
