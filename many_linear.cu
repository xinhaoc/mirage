#include "hip/hip_runtime.h"
// #include "include/mirage/persistent_kernel/tasks/linear.cuh"
#include "include/mirage/persistent_kernel/tasks/linear_cutlass.cuh"
#include "include/mirage/persistent_kernel/tasks/linear_cutlass_split.cuh"

static constexpr int SINGLE_KERNEL_THREADS = 128;
static constexpr int MAX_SHARE_MEMORY_SIZE = 160 * 1024;
static constexpr size_t NUM_LAYERS = 30;
static constexpr size_t SM_COUNT = 96;
static constexpr size_t OUTPUT_SIZE = 64;
static constexpr size_t REDUCTION_SIZE = 4096;
static constexpr size_t BATCH_SIZE = 16;
static constexpr bool USE_PIPELINE = true;
using bfloat16 = type::bfloat16_t;

__global__ void main_kernel(void *d_input, void *d_weight, void *d_output, size_t *clock_cycles_mem, size_t *clock_cycles_compute) {
    extern __shared__ char smem[];
  
    if constexpr (USE_PIPELINE) {
      kernel::linear_prefetch<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(d_input, d_weight, smem);
      for (size_t layer_num = 0; layer_num < NUM_LAYERS; layer_num++) {
        char * shared_mem_start = smem + (MAX_SHARE_MEMORY_SIZE / 2) * (layer_num % 2);

        void * input_ptr = (bfloat16 *)d_input + (layer_num * BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr = (bfloat16 *)d_weight + (layer_num * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);
        void * output_ptr = (bfloat16 *)d_output + (layer_num * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);

        char * smem_next = smem + (MAX_SHARE_MEMORY_SIZE / 2) * ((layer_num + 1) % 2);
        void * input_ptr_next = (bfloat16 *)d_input + ((layer_num + 1) * BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr_next = (bfloat16 *)d_weight + ((layer_num + 1) * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);

        kernel::linear_main<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(
        input_ptr,
        weight_ptr,
        nullptr,
        output_ptr,
        BATCH_SIZE,
        false,
        shared_mem_start,
        layer_num < NUM_LAYERS - 1,
        input_ptr_next,
        weight_ptr_next,
        smem_next,
        clock_cycles_mem + layer_num * (REDUCTION_SIZE / 128),
        clock_cycles_compute + layer_num * (REDUCTION_SIZE / 128)
        );

        kernel::linear_prefetch<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>
      }
    }

    else {
      for (size_t layer_num = 0; layer_num < NUM_LAYERS; layer_num++) {
        void * input_ptr = (bfloat16 *)d_input + (layer_num * BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr = (bfloat16 *)d_weight + (layer_num * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);
        void * output_ptr = (bfloat16 *)d_output + (layer_num * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);

        kernel::linear_kernel<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(
        input_ptr,
        weight_ptr,
        nullptr,
        output_ptr,
        BATCH_SIZE,
        false,
        clock_cycles_mem + layer_num * (REDUCTION_SIZE / 128),
        clock_cycles_compute + layer_num * (REDUCTION_SIZE / 128)
        );
      }
    }

}


int main() {

  // Create synthetic inputs and weight tensors, hipMemcpy to device memory


  // Launch the main kernel and start the timer

  int device;
  hipGetDevice(&device);
  int sm_count;
  hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);
  printf("a single persistent kernel\n");

  // Allocate device memory for d_input, d_weight, d_output and fill with ones

  size_t input_size = NUM_LAYERS * BATCH_SIZE * REDUCTION_SIZE * sizeof(bfloat16);
  size_t weight_size = NUM_LAYERS * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT * sizeof(bfloat16);
  size_t output_size = NUM_LAYERS * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT * sizeof(bfloat16);

  bfloat16 *d_input = nullptr;
  bfloat16 *d_weight = nullptr;
  bfloat16 *d_output = nullptr;

  hipMalloc(&d_input, input_size);
  hipMalloc(&d_weight, weight_size);
  hipMalloc(&d_output, output_size);

  // Fill with ones
  // Allocate host buffers
  bfloat16 *h_input = (bfloat16*)malloc(input_size);
  bfloat16 *h_weight = (bfloat16*)malloc(weight_size);
  bfloat16 *h_output = (bfloat16*)malloc(output_size);

  for (size_t i = 0; i < input_size / sizeof(bfloat16); ++i) {
      h_input[i] = bfloat16(1.0f);
  }
  for (size_t i = 0; i < weight_size / sizeof(bfloat16); ++i) {
      h_weight[i] = bfloat16(1.0f);
  }
  for (size_t i = 0; i < output_size / sizeof(bfloat16); ++i) {
      h_output[i] = bfloat16(1.0f);
  }

  hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, weight_size, hipMemcpyHostToDevice);
  hipMemcpy(d_output, h_output, output_size, hipMemcpyHostToDevice);

  free(h_input);
  free(h_weight);

  // Allocate device memory for clock_cycles_mem and clock_cycles_compute
  size_t *d_clock_cycles_mem = nullptr;
  size_t *d_clock_cycles_compute = nullptr;
  hipMalloc(&d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  hipMalloc(&d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
    
  // Launcher persistent kernel
  hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel),
                        hipFuncAttributeMaxDynamicSharedMemorySize,
                        MAX_SHARE_MEMORY_SIZE);
  

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  main_kernel<<<dim3(sm_count, 1, 1),
                      dim3(SINGLE_KERNEL_THREADS, 1, 1),
                      MAX_SHARE_MEMORY_SIZE /*smem*/>>>(d_input, d_weight, d_output, d_clock_cycles_mem, d_clock_cycles_compute);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }

  // Stop the timer and print the time
  float elapsed_ms;
  hipEventElapsedTime(&elapsed_ms, start, stop);
  printf("Time taken: %f ms\n", elapsed_ms);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Output the output tensors to a file for verification
  hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < output_size / sizeof(bfloat16); ++i) {
    if (h_output[i] != static_cast<bfloat16>(REDUCTION_SIZE)) {
      printf("Error: h_output[%zu] = %f\n", i, float(h_output[i]));
      return 1;
    }
  }

  // Write the clock_cycles_mem and clock_cycles_compute to a file
  size_t *h_clock_cycles_mem = (size_t*)malloc(NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  size_t *h_clock_cycles_compute = (size_t*)malloc(NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  hipMemcpy(h_clock_cycles_mem, d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t), hipMemcpyDeviceToHost);
  hipMemcpy(h_clock_cycles_compute, d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < NUM_LAYERS * (REDUCTION_SIZE / 128); ++i) {
    printf("clock_cycles_mem[%zu] = %zu\n", i, h_clock_cycles_mem[i]);
    printf("clock_cycles_compute[%zu] = %zu\n", i, h_clock_cycles_compute[i]);
  }
  free(h_clock_cycles_mem);
  free(h_clock_cycles_compute);


}