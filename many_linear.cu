#include "hip/hip_runtime.h"
// #include "include/mirage/persistent_kernel/tasks/linear.cuh"
#define MEASURE 0
#include "include/mirage/persistent_kernel/tasks/linear_cutlass.cuh"
#include "include/mirage/persistent_kernel/tasks/linear_cutlass_split.cuh"
#include <vector>
#include <array>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>

static constexpr int SINGLE_KERNEL_THREADS = 128;
static constexpr int MAX_SHARE_MEMORY_SIZE = 160 * 1024;
static constexpr size_t NUM_LAYERS = 30;
static constexpr size_t SM_COUNT = 96;
static constexpr size_t OUTPUT_SIZE = 64;
static constexpr size_t REDUCTION_SIZE = 1024;
static constexpr size_t BATCH_SIZE = 16;

static constexpr bool USE_PIPELINE = true;
static constexpr size_t NUM_TRIALS = 100;
static constexpr size_t NUM_WARMUP_TRIALS = 5;
#define USE_DRIVER 0
using bfloat16 = type::bfloat16_t;        // kernel::linear_prefetch<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(input_ptr_next, weight_ptr_next, smem_next);
#define CU_CHECK(err) do { if (err != hipSuccess) { printf("CU error: %d\n", err); return 1; } } while (0)
#define CUDA_CHECK(err) do { if (err != hipSuccess) { printf("CUDA error: %s\n", hipGetErrorString(err)); return 1; } } while (0)

__global__ void main_kernel(void *d_input, void *d_weight, void *d_output, size_t *clock_cycles_mem, size_t *clock_cycles_compute) {
    extern __shared__ char smem[];
  
    if constexpr (USE_PIPELINE) {
      size_t time_start_prefetch = clock64();
      kernel::linear_prefetch<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(d_input, d_weight, smem);
      size_t time_end_prefetch = clock64();
      char * shared_mem_start, * smem_next;
      shared_mem_start = smem;
      smem_next = smem + (MAX_SHARE_MEMORY_SIZE / 2);

      for (size_t layer_num = 0; layer_num < NUM_LAYERS; layer_num++) {
        // char * shared_mem_start = smem + (MAX_SHARE_MEMORY_SIZE / 2) * (layer_num % 2);

        size_t block_idx = blockIdx.x;
        void * input_ptr = (bfloat16 *)d_input + (layer_num * BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr = (bfloat16 *)d_weight + (layer_num * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (block_idx * OUTPUT_SIZE);
        void * output_ptr = (bfloat16 *)d_output + (layer_num * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT) + (block_idx * OUTPUT_SIZE);

        // char * smem_next = smem + (MAX_SHARE_MEMORY_SIZE / 2) * ((layer_num + 1) % 2);
        // void * input_ptr_next = (bfloat16 *)d_input + ((layer_num + 1) * BATCH_SIZE * REDUCTION_SIZE);
        // void * weight_ptr_next = (bfloat16 *)d_weight + ((layer_num + 1) * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);
        void * input_ptr_next = (char *) input_ptr + (BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr_next = (char*) weight_ptr + (REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT);

        kernel::linear_main<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(
        input_ptr,
        weight_ptr,
        nullptr,
        output_ptr,
        BATCH_SIZE,
        false,
        shared_mem_start,
        layer_num < NUM_LAYERS - 1,
        smem_next,
        input_ptr_next,
        weight_ptr_next
        );
        
        char* temp;
        temp = shared_mem_start;
        shared_mem_start = smem_next;
        smem_next = temp;

      }
    }

    else {
      for (size_t layer_num = 0; layer_num < NUM_LAYERS; layer_num++) {
        void * input_ptr = (bfloat16 *)d_input + (layer_num * BATCH_SIZE * REDUCTION_SIZE);
        void * weight_ptr = (bfloat16 *)d_weight + (layer_num * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);
        void * output_ptr = (bfloat16 *)d_output + (layer_num * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT) + (blockIdx.x * OUTPUT_SIZE);

        kernel::linear_kernel<bfloat16, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE, OUTPUT_SIZE * SM_COUNT>(
        input_ptr,
        weight_ptr,
        nullptr,
        output_ptr,
        BATCH_SIZE,
        false
        );
      }
    }

}


int main() {

  // Create synthetic inputs and weight tensors, hipMemcpy to device memory


  // Launch the main kernel and start the timer
  hipSetDevice(6);

  int device;
  hipGetDevice(&device);
  int sm_count;
  hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);
  printf("a single persistent kernel\n");

#if USE_DRIVER
  hipDevice_t cu_device;
  hipCtx_t cu_context;
  hipModule_t mod;
  hipFunction_t main_kernel;
  hipInit(0);
  hipDeviceGet(&cu_device, 0);
  hipCtxCreate(&cu_context, 0, cu_device);
  auto result = hipModuleLoad(&mod, "many_linear.ptx");
  if (result != hipSuccess) {
    printf("Error loading module: %d\n", result);
    return 1;
  }
  auto result2 = hipModuleGetFunction(&main_kernel, mod, "_Z11main_kernelPvS_S_PmS0_");
  if (result2 != hipSuccess) {
    printf("Error getting function: %d\n", result2);
    return 1;
  }
#endif

  // Allocate device memory for d_input, d_weight, d_output and fill with ones

  size_t input_size = NUM_LAYERS * BATCH_SIZE * REDUCTION_SIZE * sizeof(bfloat16);
  size_t weight_size = NUM_LAYERS * REDUCTION_SIZE * OUTPUT_SIZE * SM_COUNT * sizeof(bfloat16);
  size_t output_size = NUM_LAYERS * BATCH_SIZE * OUTPUT_SIZE * SM_COUNT * sizeof(bfloat16);

#if USE_DRIVER
  hipDeviceptr_t d_input = 0;
  hipDeviceptr_t d_weight = 0;
  hipDeviceptr_t d_output = 0;
#else
  bfloat16 *d_input = nullptr;
  bfloat16 *d_weight = nullptr;
  bfloat16 *d_output = nullptr;
#endif

#if USE_DRIVER
  CU_CHECK(hipMalloc(&d_input, input_size));
  CU_CHECK(hipMalloc(&d_weight, weight_size));
  CU_CHECK(hipMalloc(&d_output, output_size));
#else
  hipMalloc(&d_input, input_size);
  hipMalloc(&d_weight, weight_size);
  hipMalloc(&d_output, output_size);
#endif

  // Fill with ones
  // Allocate host buffers
  bfloat16 *h_input = (bfloat16*)malloc(input_size);
  bfloat16 *h_weight = (bfloat16*)malloc(weight_size);
  bfloat16 *h_output = (bfloat16*)malloc(output_size);

  for (size_t i = 0; i < input_size / sizeof(bfloat16); ++i) {
      h_input[i] = bfloat16(1.0f);
  }
  for (size_t i = 0; i < weight_size / sizeof(bfloat16); ++i) {
      h_weight[i] = bfloat16(1.0f);
  }
  for (size_t i = 0; i < output_size / sizeof(bfloat16); ++i) {
      h_output[i] = bfloat16(1.0f);
  }

#if USE_DRIVER
  CU_CHECK(hipMemcpyHtoD(d_input, h_input, input_size));
  CU_CHECK(hipMemcpyHtoD(d_weight, h_weight, weight_size));
  CU_CHECK(hipMemcpyHtoD(d_output, h_output, output_size));
#else
  hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, weight_size, hipMemcpyHostToDevice);
  hipMemcpy(d_output, h_output, output_size, hipMemcpyHostToDevice);
#endif

  free(h_input);
  free(h_weight);

  // Allocate device memory for clock_cycles_mem and clock_cycles_compute
#if USE_DRIVER
  hipDeviceptr_t d_clock_cycles_mem = 0;
  hipDeviceptr_t d_clock_cycles_compute = 0;
  CU_CHECK(hipMalloc(&d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t)));
  CU_CHECK(hipMalloc(&d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t)));
#else
  size_t *d_clock_cycles_mem = nullptr;
  size_t *d_clock_cycles_compute = nullptr;
  hipMalloc(&d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  hipMalloc(&d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
#endif
  // // Launcher persistent kernel
#if USE_DRIVER
  CU_CHECK(cuFuncSetAttribute(main_kernel, HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES, MAX_SHARE_MEMORY_SIZE));
#else
  hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, MAX_SHARE_MEMORY_SIZE);
#endif

  void* args[] = {&d_input, &d_weight, &d_output, &d_clock_cycles_mem, &d_clock_cycles_compute};
  for (size_t i = 0; i < NUM_WARMUP_TRIALS; ++i) {
  #if USE_DRIVER
    auto result = hipModuleLaunchKernel(main_kernel, sm_count, 1, 1, SINGLE_KERNEL_THREADS, 1, 1, MAX_SHARE_MEMORY_SIZE, 0, args, nullptr);
    if (result != hipSuccess) {
      printf("Error launching kernel: %d\n", result);
      return 1;
    }
  #else
    main_kernel<<<dim3(sm_count, 1, 1),
                      dim3(SINGLE_KERNEL_THREADS, 1, 1),
                      MAX_SHARE_MEMORY_SIZE /*smem*/>>>(d_input, d_weight, d_output, d_clock_cycles_mem, d_clock_cycles_compute);
  #endif
  }
  printf("Finished warmup\n");
  CUDA_CHECK(hipDeviceSynchronize());
  std::array<float, NUM_TRIALS> all_elapsed_ms;
  for (size_t i = 0; i < NUM_TRIALS; ++i) {

    #if USE_DRIVER
    hipEvent_t start, stop;
    CU_CHECK(hipEventCreateWithFlags(&start, hipEventDefault));
    CU_CHECK(hipEventCreateWithFlags(&stop, hipEventDefault));
    CU_CHECK(hipEventRecord(start, 0));
    auto result = hipModuleLaunchKernel(main_kernel, sm_count, 1, 1, SINGLE_KERNEL_THREADS, 1, 1, MAX_SHARE_MEMORY_SIZE, 0, args, nullptr);
    CU_CHECK(hipEventRecord(stop, 0));
    CU_CHECK(hipEventSynchronize(stop));
    float elapsed_ms;
    CU_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
    all_elapsed_ms[i] = elapsed_ms;
    if (result != hipSuccess) {
      printf("Error launching kernel: %d\n", result);
      return 1;
    }
    CU_CHECK(hipEventDestroy(start));
    CU_CHECK(hipEventDestroy(stop));
    #else
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    main_kernel<<<dim3(sm_count, 1, 1),
                      dim3(SINGLE_KERNEL_THREADS, 1, 1),
                      MAX_SHARE_MEMORY_SIZE /*smem*/>>>(d_input, d_weight, d_output, d_clock_cycles_mem, d_clock_cycles_compute);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    all_elapsed_ms[i] = elapsed_ms;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif
  }
  printf("Finished trials\n");
  CUDA_CHECK(hipDeviceSynchronize());
  // Process the elapsed times
  float min_elapsed_ms = *std::min_element(all_elapsed_ms.begin(), all_elapsed_ms.end());
  float max_elapsed_ms = *std::max_element(all_elapsed_ms.begin(), all_elapsed_ms.end());
  float average_elapsed_ms = std::accumulate(all_elapsed_ms.begin(), all_elapsed_ms.end(), 0.0f) / NUM_TRIALS;
  float std_elapsed_ms = std::sqrt(std::accumulate(all_elapsed_ms.begin(), all_elapsed_ms.end(), 0.0f, [average_elapsed_ms](float acc, float x) { return acc + (x - average_elapsed_ms) * (x - average_elapsed_ms); }) / NUM_TRIALS);
  printf("Min elapsed time: %f ms\n", min_elapsed_ms);
  printf("Max elapsed time: %f ms\n", max_elapsed_ms);
  printf("Average elapsed time: %f ms\n", average_elapsed_ms);
  printf("Standard deviation: %f ms (%.2f%%)\n", std_elapsed_ms, std_elapsed_ms / average_elapsed_ms * 100);
  // print all the elapsed times
  for (size_t i = 0; i < NUM_TRIALS; ++i) {
    printf("Elapsed time %zu: %f ms, ", i, all_elapsed_ms[i]);
  }

  // Output the output tensors to a file for verification
  #if USE_DRIVER
  hipMemcpyDtoH(h_output, d_output, output_size);
  #else
  hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);
  #endif
  for (size_t i = 0; i < output_size / sizeof(bfloat16); ++i) {
    if (h_output[i] != static_cast<bfloat16>(REDUCTION_SIZE)) {
      printf("Error: h_output[%zu] = %f\n", i, float(h_output[i]));
      return 1;
    }
  }

  // Write the clock_cycles_mem and clock_cycles_compute to a file
  size_t *h_clock_cycles_mem = (size_t*)malloc(NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  size_t *h_clock_cycles_compute = (size_t*)malloc(NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t));
  #if USE_DRIVER
  CU_CHECK(hipMemcpyDtoH(h_clock_cycles_mem, d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t)));
  CU_CHECK(hipMemcpyDtoH(h_clock_cycles_compute, d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t)));
  #else
  hipMemcpy(h_clock_cycles_mem, d_clock_cycles_mem, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t), hipMemcpyDeviceToHost);
  hipMemcpy(h_clock_cycles_compute, d_clock_cycles_compute, NUM_LAYERS * (REDUCTION_SIZE / 128) * sizeof(size_t), hipMemcpyDeviceToHost);
  #endif
  for (size_t i = 0; i < NUM_LAYERS * (REDUCTION_SIZE / 128); ++i) {
    printf("clock_cycles_mem[%zu] = %zu\n", i, h_clock_cycles_mem[i]);
    printf("clock_cycles_compute[%zu] = %zu\n", i, h_clock_cycles_compute[i]);
  }
  free(h_clock_cycles_mem);
  free(h_clock_cycles_compute);

#if USE_DRIVER
  hipCtxDestroy(cu_context);
#endif
}